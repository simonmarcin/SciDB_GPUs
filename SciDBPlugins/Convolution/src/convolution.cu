#include "hip/hip_runtime.h"
/**
 * @file convolution.cu
 *
 * @author simon.marcin
 *
 * @brief Messy convolution algorithm to test all possible designs and parameters to
 * evaluate the best setup for GPU accelerated Operators.
 * Can run synchronous copys or asynchronous, can run multiple streams or not.
 * Writes a lot of debug messages.
 *
 *
 */

#include <stdio.h>
#include "convolution.h"
#include <iostream>
#include <float.h>

#define DSIZE 1024
#define DVAL 10
#define nTPB 256
#define hipHostRegisterPortable 0x01
#define ROWS_BLOCKDIM_X 16
#define ROWS_BLOCKDIM_Y 16


#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)



template <class type> __global__ void convolutionKernel(float* d_Output,type* d_Input, size_t chunckWidth,
		int overlap, size_t kernelSize, float* d_Kernel, int repetitions)
{
	// define shared memory
	const int globalWidth = chunckWidth+(2*overlap);
    //__shared__ float s_Data[34][34];
    extern __shared__ float s_Data[];

    // offset to overlaps (halo elements)
    const int globalX = (blockIdx.x * ROWS_BLOCKDIM_X) + threadIdx.x;
    const int globalY = (blockIdx.y * ROWS_BLOCKDIM_Y) + threadIdx.y;

    // load inner chunk elements
    s_Data[(threadIdx.y+overlap)*(ROWS_BLOCKDIM_X+2*overlap) + threadIdx.x+overlap] = d_Input[((globalY+overlap)*globalWidth)+globalX+overlap];
    //printf("thread %d,%d, input=%d to=%d,%d\n", threadIdx.y,threadIdx.x, d_Input[((globalY+overlap)*globalWidth)+globalX+overlap], threadIdx.y+overlap,threadIdx.x+overlap);

    // // load upper halo elements
    if (threadIdx.y < overlap){
    	s_Data[(threadIdx.y)*(ROWS_BLOCKDIM_X+2*overlap) + threadIdx.x+overlap]=d_Input[globalY*globalWidth + globalX+overlap];
    }
    //load bottom halo elements
    if (threadIdx.y >= ROWS_BLOCKDIM_Y-overlap){
    	s_Data[(threadIdx.y+2*overlap)*(ROWS_BLOCKDIM_X+2*overlap) + threadIdx.x+overlap]=d_Input[(globalY+2*overlap)*globalWidth + globalX+overlap];
    }
    // load left halo elements
    if (threadIdx.x < overlap){
    	s_Data[(threadIdx.y+overlap)*(ROWS_BLOCKDIM_X+2*overlap) + threadIdx.x]=d_Input[(globalY+overlap)*globalWidth + globalX];
    }
    // load right halo elements
    if (threadIdx.x >= ROWS_BLOCKDIM_X-overlap){
    	s_Data[(threadIdx.y+overlap)*(ROWS_BLOCKDIM_X+2*overlap) + threadIdx.x+2*overlap]=d_Input[(globalY+overlap)*globalWidth + globalX+2*overlap];
    }

    // left upper edges
    if (threadIdx.x < overlap and threadIdx.y < overlap){
    	s_Data[(threadIdx.y)*(ROWS_BLOCKDIM_X+2*overlap) + threadIdx.x]=d_Input[(globalY)*globalWidth + globalX];
    }
    // right upper edges
    if (threadIdx.x >= ROWS_BLOCKDIM_X-overlap and threadIdx.y < overlap){
    	s_Data[(threadIdx.y)*(ROWS_BLOCKDIM_X+2*overlap) + threadIdx.x+2*overlap]=d_Input[(globalY)*globalWidth + globalX+2*overlap];
    }
    // left bottom edges
    if (threadIdx.x < overlap and threadIdx.y >= ROWS_BLOCKDIM_Y-overlap){
    	s_Data[(threadIdx.y+2*overlap)*(ROWS_BLOCKDIM_X+2*overlap) + threadIdx.x]=d_Input[(globalY+2*overlap)*globalWidth + globalX];
    }
    // right bottom edges
    if (threadIdx.x >= ROWS_BLOCKDIM_X-overlap and threadIdx.y >= ROWS_BLOCKDIM_Y-overlap){
    	s_Data[(threadIdx.y+2*overlap)*(ROWS_BLOCKDIM_X+2*overlap) + threadIdx.x+overlap]=d_Input[(globalY+2*overlap)*globalWidth + globalX+2*overlap];
    }

    //Compute and store results
    __syncthreads();


    float sum = 0;

    for (int reps=0;reps<repetitions;reps++) {

		#pragma unroll
		for (int j = -overlap; j <= overlap; j++){
			#pragma unroll
			for (int i = -overlap; i <= overlap; i++){
				//sum += s_Data[(threadIdx.y+j+overlap)*(ROWS_BLOCKDIM_X+2*overlap) + threadIdx.x+i+overlap];
				sum += d_Kernel[ ((int)kernelSize/2)  +(j*( 2*overlap+1 ))+i] * s_Data[(threadIdx.y+j+overlap)*(ROWS_BLOCKDIM_X+2*overlap) +  threadIdx.x+i+overlap];
			}
		}
    }

	d_Output[globalY*chunckWidth+globalX] = sum;

}

// init the template function as the CUDA compiler doesn't know anything about SciDB
template __global__ void convolutionKernel<int16_t>(float*,int16_t*, size_t,int, size_t, float*, int);
template __global__ void convolutionKernel<int32_t>(float*,int32_t*, size_t,int, size_t, float*, int);


template <class type> void GPUHandle<type>::copyChunkValues(int i){

	// transfer data to device
	if(debug){LOG4CXX_INFO(logger, "start memory transfer!");}

	if(useStreams){
		if(async){
			hipMemcpyAsync(d_Input[i], chunkValues[i], valueBytes, hipMemcpyHostToDevice,(streams[i]));
			cudaCheckErrors("hipMemcpyAsync (async): chunkValues");
		}else{
			hipMemcpyAsync(d_Input[0], chunkValues[0], valueBytes, hipMemcpyHostToDevice,(streams[0]));
			cudaCheckErrors("hipMemcpyAsync (streams): chunkValues");
		}
	}else{
		hipMemcpy(d_Input[0], chunkValues[0], valueBytes, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		cudaCheckErrors("hipMemcpy: chunkValues");
	}

    if(debug){LOG4CXX_INFO(logger, "memory transfer done!");}

}

template <class type> void GPUHandle<type>::waitEvent(int i){
	hipEventSynchronize(syncEvents[i]);
	cudaCheckErrors("hipEventSynchronize");
}

template <class type> void GPUHandle<type>::copyResultValues(int i){

	// transfer data to host
	if(debug){LOG4CXX_INFO(logger, "start memory transfer!");}
	if(useStreams){
		if(async){
			hipMemcpyAsync(resultValues[i], d_Output[i], resultBytes, hipMemcpyDeviceToHost, (streams[i]));
			// Inject a sync Event as we need to be sure that we have the result
			hipEventRecord(syncEvents[i],streams[i]);
			cudaCheckErrors("hipMemcpyAsync (async): d_Output");
		}else{
			hipMemcpyAsync(resultValues[0], d_Output[0], resultBytes, hipMemcpyDeviceToHost, (streams[0]));
			hipStreamSynchronize((streams[0]));
			cudaCheckErrors("hipMemcpyAsync (streams): d_Output");
		}
	}else{
		hipMemcpy(resultValues[0], d_Output[0], resultBytes, hipMemcpyDeviceToHost);
		cudaCheckErrors("hipMemcpy: d_Output");
	}
	if(debug){LOG4CXX_INFO(logger, "memory transfer done!");}

}

template <class type> void GPUHandle<type>::runConvolution(int i, int repetitions){

	// define the kernel grid (2D)

    dim3 blocks(max(chunckWidth / ROWS_BLOCKDIM_X,size_t(1)), max(chunckWidth / ROWS_BLOCKDIM_Y,size_t(1)));
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

   // run kernel
    if(debug){LOG4CXX_INFO(logger, chunckWidth<< "  "<<overlap<< "  "<<kernelSize);}
    if(useStreams){
    	if(async){
    		convolutionKernel<<<blocks, threads, smBytes, (streams[i])>>>(d_Output[i],d_Input[i],chunckWidth,overlap,kernelSize, d_Kernel, repetitions);
			cudaCheckErrors("convolutionRowsKernel() (streams)");
    	}else{
    		convolutionKernel<<<blocks, threads, smBytes, (streams[0])>>>(d_Output[0],d_Input[0],chunckWidth,overlap,kernelSize, d_Kernel, repetitions);
			cudaCheckErrors("convolutionRowsKernel() (streams)");
    	}
    }else{
    	if(debug){LOG4CXX_INFO(logger, "blocks: " <<  chunckWidth / ROWS_BLOCKDIM_X << "," << chunckWidth / ROWS_BLOCKDIM_Y << "," << smBytes);}
		convolutionKernel<<<blocks, threads, smBytes>>>(d_Output[0],d_Input[0],chunckWidth,overlap,kernelSize, d_Kernel, repetitions);
		hipDeviceSynchronize();
		cudaCheckErrors("convolutionRowsKernel()");
    }
    if(debug){LOG4CXX_INFO(logger, "kernel done!");}



}

template <class type> GPUHandle<type>::~GPUHandle(){

	// unpin memory
	hipHostUnregister(chunkValues[0]);
	cudaCheckErrors("cudaHostUnRegister: chunkValues");
	hipHostUnregister(resultValues[0]);
	cudaCheckErrors("cudaHostUnRegister: resultValues");
	if(async){
		hipHostUnregister(chunkValues[1]);
		cudaCheckErrors("cudaHostUnRegister: chunkValues[1]");
		hipHostUnregister(resultValues[1]);
		cudaCheckErrors("cudaHostUnRegister: resultValues[1]");
	}

	// free memory on device
	hipFree(d_Input[0]);
	cudaCheckErrors("hipFree: d_Input");
	hipFree(d_Output[0]);
	cudaCheckErrors("hipFree: d_Output");
	hipFree(d_Kernel);
	cudaCheckErrors("hipFree: d_Kernel");
	if(async){
		hipFree(d_Input[1]);
		cudaCheckErrors("hipFree async: d_Input");
		hipFree(d_Output[1]);
		cudaCheckErrors("hipFree async: d_Output");
	}

	// destroy streams
	if(useStreams){
		hipStreamDestroy(streams[0]);
		hipStreamDestroy(streams[1]);
	}

	// destroy sync events
	if(async){
		hipEventDestroy(syncEvents[0]);
		cudaCheckErrors("hipEventDestroy: syncEvents[0]");
		hipEventDestroy(syncEvents[1]);
		cudaCheckErrors("hipEventDestroy: syncEvents[1]");
	}
}


template <class type> GPUHandle<type>::GPUHandle(float* in_kernel, size_t in_kernelBytes,type* in_chunkValues[],
		size_t in_valueBytes, float* in_resultValues[], size_t in_resultBytes,
		log4cxx::LoggerPtr in_logger, size_t in_chunckWidth, size_t in_kernelSize, bool in_debug,
		bool in_useStreams, bool in_async){

	// save values
	kernel = in_kernel;
	kernelBytes = in_kernelBytes;
	chunkValues[0] = in_chunkValues[0];
	chunkValues[1] = in_chunkValues[1];
	valueBytes = in_valueBytes;
	resultValues[0] = in_resultValues[0];
	resultValues[1] = in_resultValues[1];
	resultBytes = in_resultBytes;
	logger = in_logger;
	overlap = (int) sqrt(in_kernelSize)/2;
	kernelSize = in_kernelSize;
	chunckWidth = in_chunckWidth;
	smBytes = (ROWS_BLOCKDIM_X+2*overlap)*(ROWS_BLOCKDIM_Y+2*overlap)*sizeof(float);
	debug = in_debug;
	useStreams = in_useStreams;
	async = in_async;

	// create streams
	if(useStreams){
		streams[0] = hipStream_t();
		hipStreamCreateWithFlags(&(streams[0]),hipStreamNonBlocking);
		cudaCheckErrors("hipStreamCreateWithFlags: streams[0]");
		streams[1] = hipStream_t();
		hipStreamCreateWithFlags(&(streams[1]),hipStreamNonBlocking);
		cudaCheckErrors("hipStreamCreateWithFlags: streams[1]");
	}

	// create sync events
	if(async){
		syncEvents[0] = hipEvent_t();
		syncEvents[1] = hipEvent_t();
		hipEventCreateWithFlags(&(syncEvents[0]), hipEventDisableTiming);
		cudaCheckErrors("hipEventCreateWithFlags: syncEvents[0]");
		hipEventCreateWithFlags(&(syncEvents[1]), hipEventDisableTiming);
		cudaCheckErrors("hipEventCreateWithFlags: syncEvents[1]");
	}

	// pin host memory (this is slow but we only do it once to allow async transfers)
	hipHostRegister(chunkValues[0], valueBytes, hipHostRegisterPortable);
	cudaCheckErrors("hipHostRegister: chunkValues[0]");
	hipHostRegister(resultValues[0], resultBytes, hipHostRegisterPortable);
	cudaCheckErrors("hipHostRegister: resultValues[0]");
	if(async){
		hipHostRegister(chunkValues[1], valueBytes, hipHostRegisterPortable);
		cudaCheckErrors("hipHostRegister: chunkValues[1]");
		hipHostRegister(resultValues[1], resultBytes, hipHostRegisterPortable);
		cudaCheckErrors("hipHostRegister: resultValues[1]");
	}

	// prepare Memory on device
	hipMalloc((void **)&(d_Input[0]), valueBytes);
	cudaCheckErrors("hipMalloc: d_Input");
	hipMalloc((void **)&(d_Output[0]), resultBytes);
	cudaCheckErrors("hipMalloc: d_Output");
	hipMalloc((void **)&d_Kernel, kernelBytes);
	cudaCheckErrors("hipMalloc: d_Kernel");
	if(async){
		hipMalloc((void **)&(d_Input[1]), valueBytes);
		cudaCheckErrors("hipMalloc async: d_Input");
		hipMalloc((void **)&(d_Output[1]), resultBytes);
		cudaCheckErrors("hipMalloc async: d_Output");
	}

	// copy kernel to constant memory
	//hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), kernel, kernelBytes);

	// copy kernel to global memory (as the size is not known at compilation time)
	if(useStreams){
		hipMemcpyAsync(d_Kernel, kernel, kernelBytes, hipMemcpyHostToDevice, streams[0]);
	    cudaCheckErrors("hipMemcpyAsync (streams): d_Kernel");
	}else{
	    hipMemcpy(d_Kernel, kernel, kernelBytes, hipMemcpyHostToDevice);
	    cudaCheckErrors("hipMemcpy: d_Kernel");
	}

}

// init the template classes as nvcc doesn't know the used types
template class GPUHandle<int16_t>;
template class GPUHandle<int32_t>;


